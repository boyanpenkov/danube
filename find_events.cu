#include "hip/hip_runtime.h"
#define MACRO_THREADS 1024
#define NOT_EVENT 0
#define INTERVAL 500
#define USE_NVTX

#ifdef USE_NVTX
#include "nvToolsExt.h"

const uint32_t colors[] = { 0xff00ff00, 0xff0000ff, 0xffffff00, 0xffff00ff, 0xff00ffff, 0xffff0000, 0xffffffff };
const int num_colors = sizeof(colors)/sizeof(uint32_t);

#define PUSH_RANGE(name,cid) { \
    int color_id = cid; \
    color_id = color_id%num_colors;\
    nvtxEventAttributes_t eventAttrib = {0}; \
    eventAttrib.version = NVTX_VERSION; \
    eventAttrib.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE; \
    eventAttrib.colorType = NVTX_COLOR_ARGB; \
    eventAttrib.color = colors[color_id]; \
    eventAttrib.messageType = NVTX_MESSAGE_TYPE_ASCII; \
    eventAttrib.message.ascii = name; \
    nvtxRangePushEx(&eventAttrib); \
}
#define POP_RANGE nvtxRangePop();
#else
#define PUSH_RANGE(name,cid)
#define POP_RANGE
#endif

#define PPT 64
#define FILT_WINDOW 5
#define EVENTS 5

//1 = text file
//2 = binary file
#define READ_OPT 2

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>
#include <fcntl.h>
#include <assert.h>
#include <string.h>

#include <fstream>
#include <iostream>

int SIZE; // length of the data file going in, in units of points...

#include "find_transitions_mean.h"
#include "find_transitions_delta.h"
#include "find_transitions_canny.h"
#include "find_transitions_c.h"
#include "find_high_random.h"
#include "mean_filter_signal.h"

float *h_values;
float *h_values_raw;
float *h_high_mean;
float *d_values;
int *d_transitions;
float *d_high_mean;
int *d_size;
float *d_gradient;
float *d_smoothed;

int main(int argc, char ** argv) {

  int expected_values = EVENTS;
  int inBinaryFile;

  const char *available_kernels[4];
  available_kernels[0] = "mean";
  available_kernels[1] = "delta";
  available_kernels[2] = "canny";
  available_kernels[3] = "c";

  if (READ_OPT == 1)
    {
      printf("You are reading the CSV as an arg!\n");
      char *filename = strdup(argv[2]);
      std::ifstream inTextFile;
      // Open the file and read data
      inTextFile.open(filename);
      if (!inTextFile)
	{
	  printf("\nFailed to open file on csv run!\n");
	  return 1;
	}
      // Read SIZE from first line of the file
      inTextFile >> SIZE;
    }
  else
    {
      char filename[] = "signal.dat";
      if ((inBinaryFile = open(filename, O_RDONLY)) < 0)
	{
	  printf("\nFailed to open the file on dat run!\n");
	  return 1;
	}
      float *lines;
      lines = (float*)calloc(1, sizeof(float));
      read(inBinaryFile, lines, sizeof(float));
      SIZE = (*lines);
    }

  assert(SIZE > 0);
  printf("%d was the number of points you just passed in.\n", SIZE);

  const int THREADS = MACRO_THREADS;
  const int BLOCKS = floor((((float)SIZE / (float)THREADS)) / PPT);
  printf("Block count: %d.\n", BLOCKS);
  fflush(stdout);
  const int cropped_size = BLOCKS*THREADS*PPT;
  const int cropped_bytes = cropped_size*sizeof(float);
  printf("%d was as close as I could get.\n", cropped_size);
  fflush(stdout);

  assert(THREADS > 0);
  assert(BLOCKS > 0);
  assert(cropped_size > 0);
  assert(cropped_size < SIZE);

  // Now, copy the input and drop the last few points

  const int ARRAY_BYTES = SIZE * sizeof(float);
  h_values_raw = (float *)calloc(SIZE, sizeof(float));
  if (READ_OPT == 1)
    {
      char *filename = strdup(argv[2]);
      std::ifstream inTextFile;
      //Open the file and read data
      inTextFile.open(filename);
      if (!inTextFile)
	{
	  printf("\nFailed to open file");
	  return 1;
	}
      printf("You are pulling data from the CSV.\n");
      for (int i = 0; i < SIZE; i++)
	inTextFile >> h_values_raw[i];
    }
  else
    {
      read(inBinaryFile, h_values_raw, ARRAY_BYTES);
    }

  hipHostMalloc((void**) &h_values, cropped_bytes);

  for(int i=1; i<cropped_size; i++)
    {
        h_values[i] = h_values_raw[i];
    }

  free(h_values_raw);

  FILE *f;  // Regardless of CPU or GPU, this is the file you're writing results to.
  h_high_mean = (float*)calloc(1, sizeof(float));

  if (argc == 1)
    {
      printf("Run with one of the arguments: ");
      for (int i = 0; i < sizeof(available_kernels) / sizeof(available_kernels[0]); i++)
	printf("%s ", available_kernels[i]);
      printf("\n");
      return 1;
    }

  if (strcmp(argv[1], "c") == 0)
    {
      PUSH_RANGE("CPU run.",1)
      printf("Using CPU.\n");
      // Now you are not using the GPU at all, and are just on C on the CPU.
      // Run the relevant transition finder, using a multipass finder for now.
      *h_high_mean = find_high_random(h_values);
      float *h_transitions = (float*)calloc(cropped_size, sizeof(float));
      int passes = 3; // How many passes do you want your multipass eventfinder to run on?
      find_transitions_c( h_values, h_transitions, *h_high_mean, passes, cropped_size);
      // This is going to modify h_values, so get rid of it for safety.
      hipFree(h_values);
      // open the correct guessed transition file for writing.
      f = fopen("transitions_guessed_c.csv", "w");
      // Write the found transitions to the correct file you opened above.
      for (int i = 0; i < cropped_size; i++)
	fprintf(f, "%f\n", h_transitions[i]);
      fclose(f);
      printf("CPU run done.\n");
      POP_RANGE
    }

  else
    { // You are in the GPU branch.
      // Allocate GPU memory
      printf("Using GPU.\n");
      hipMalloc((void**) &d_values, cropped_bytes);
      hipMalloc((void**) &d_smoothed, cropped_bytes);
      hipMalloc((void**) &d_transitions, sizeof(int) * BLOCKS);
      hipMalloc((void**) &d_high_mean, sizeof(float));
      hipMalloc((void**) &d_size, sizeof(int));
      hipMalloc((void**) &d_gradient, cropped_bytes);

      hipStream_t stream1;
      hipStreamCreate(&stream1);
      hipMemcpyAsync(d_values, h_values, cropped_bytes, hipMemcpyHostToDevice, stream1);
      printf("Host-to-device copy initiated.\n");
      fflush(stdout);
      // Launch the kernel

      printf("All pre-kernel launch stuff OK.\n");
      fflush(stdout);
      if (strcmp(argv[1], "delta") == 0)
	{
	  // Transfer the array to GPU
	  hipMemcpy(d_size, &cropped_size, sizeof(int), hipMemcpyHostToDevice);
	  // Run the relevant transition finder
	  find_transitions_delta <<< BLOCKS, THREADS, 0, stream1 >>> (d_values, d_transitions, PPT, MACRO_THREADS);
	  // open the correct guessed transition file for writing.
	  f = fopen("transitions_guessed_delta.csv", "w");
	  expected_values = EVENTS*2;
	  // copy the result back to CPU
	  hipMemcpy(h_values, d_values, cropped_bytes, hipMemcpyDeviceToHost);
	}
      else if (strcmp(argv[1], "mean") == 0)
	{
	  *h_high_mean = find_high_random(h_values);
	  hipMemcpy(d_high_mean, h_high_mean, sizeof(float), hipMemcpyHostToDevice);
	  find_transitions_mean <<< BLOCKS, THREADS, 0, stream1 >>> (d_values, d_transitions, PPT, MACRO_THREADS, d_high_mean);
	  f = fopen("transitions_guessed_mean.csv", "w");
	  hipMemcpy(h_values, d_values, cropped_bytes, hipMemcpyDeviceToHost);
	}
      else if (strcmp(argv[1], "canny") == 0)
	{
	  hipMemcpy(d_size, &cropped_size, sizeof(int), hipMemcpyHostToDevice);
	  mean_filter_signal <<< BLOCKS, THREADS,0, stream1 >>> (d_values, PPT, FILT_WINDOW, d_size, d_smoothed);
	  find_transitions_canny <<< BLOCKS, THREADS,0, stream1 >>> (d_values, d_transitions, PPT, d_size, d_gradient);
	  f = fopen("transitions_guessed_canny.csv", "w");
	  expected_values = EVENTS*2;
	  hipMemcpy(h_values, d_gradient, cropped_bytes, hipMemcpyDeviceToHost);
	}
      else
	{
	  printf("Run with one of the arguments: ");
	  for (int i = 0; i < sizeof(available_kernels) / sizeof(available_kernels[0]); i++)
	    printf("%s ", available_kernels[i]);
	  printf("\n");
	  return 1;
	}
      // free GPU memory
      hipFree(d_values);
      hipStreamDestroy(stream1);
      // Write the found transitions to the correct file you opened above.
      for (int i = 0; i < cropped_size; i++)
	fprintf(f, "%f\n", h_values[i]);
      fclose(f);
      printf("GPU run done.\n");
    }

  char eventFlag = 'F';
  int total_transitions = 0;
  for (int i = 0; i < cropped_size; i++){
    if (h_values[i] == NOT_EVENT && eventFlag == 'F'){
      continue; // you're not in an event, and you pass
	}
    else if (h_values[i] != NOT_EVENT && eventFlag == 'F'){
      eventFlag = 'T'; // walked into event
    }
    else if (h_values[i] != NOT_EVENT && eventFlag == 'T'){
      continue; // moving along event
    }
    else if (h_values[i] == NOT_EVENT && eventFlag == 'T'){
      total_transitions++; // just left an event
      eventFlag = 'F';
    }
    else { return 1;}
  }
  printf("Computed with %s : ", argv[1]);
  printf("%d (%d expected for synthetically generated data.)\n", total_transitions, expected_values);
  return 0;
}
